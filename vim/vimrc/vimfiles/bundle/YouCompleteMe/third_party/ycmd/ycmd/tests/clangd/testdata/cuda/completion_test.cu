#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Kernels
{
  __device__ void do_something(float* a) {}
}

template<typename F, class ...Args>
__global__ void launch(F& fn, Args args...)
{
  fn(args...);
}

int main() {
  // The location after the colon is line 16, col 29
  launch<<<1, 1>>>(Kernels::
  return 0;
}
