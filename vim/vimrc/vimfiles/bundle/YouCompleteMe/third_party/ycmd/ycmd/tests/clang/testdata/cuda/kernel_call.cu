#include "hip/hip_runtime.h"
/* Modified by ycmd contributors */
/*
  University of Illinois/NCSA
  Open Source License

  Copyright (c) 2007-2016 University of Illinois at Urbana-Champaign.
  All rights reserved.

  Developed by:

      LLVM Team

      University of Illinois at Urbana-Champaign

      http://llvm.org

  Permission is hereby granted, free of charge, to any person obtaining a copy of
  this software and associated documentation files (the "Software"), to deal with
  the Software without restriction, including without limitation the rights to
  use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
  of the Software, and to permit persons to whom the Software is furnished to do
  so, subject to the following conditions:

      * Redistributions of source code must retain the above copyright notice,
        this list of conditions and the following disclaimers.

      * Redistributions in binary form must reproduce the above copyright notice,
        this list of conditions and the following disclaimers in the
        documentation and/or other materials provided with the distribution.

      * Neither the names of the LLVM Team, University of Illinois at
        Urbana-Champaign, nor the names of its contributors may be used to
        endorse or promote products derived from this Software without specific
        prior written permission.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
  FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
  CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH THE
  SOFTWARE.
*/


#include "hip/hip_runtime.h"

__global__ void g1(int x) {}

template <typename T> void t1(T arg) {
  g1<<<arg, arg>>>(1);
}

void h1(int x) {}
int h2(int x) { return 1; }

int main(void) {
  g1<<<1, 1>>>(42);
  g1(42); // expected-error {{call to global function 'g1' not configured}}
  g1<<<1>>>(42); // expected-error {{too few execution configuration arguments to kernel function call}}
  g1<<<1, 1, 0, 0, 0>>>(42); // expected-error {{too many execution configuration arguments to kernel function call}}

  t1(1);

  h1<<<1, 1>>>(42); // expected-error {{kernel call to non-global function 'h1'}}

  int (*fp)(int) = h2;
  fp<<<1, 1>>>(42); // expected-error {{must have void return type}}

  g1<<<undeclared, 1>>>(42); // expected-error {{use of undeclared identifier 'undeclared'}}
}
